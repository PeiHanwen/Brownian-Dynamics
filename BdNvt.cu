#include "hip/hip_runtime.h"
/*
GALAMOST - GPU-Accelerated Large-Scale Molecular Simulation Toolkit
COPYRIGHT
	GALAMOST Copyright (c) (2013) The group of Prof. Zhong-Yuan Lu
LICENSE
	This program is a free software: you can redistribute it and/or 
	modify it under the terms of the GNU General Public License. 
	This program is distributed in the hope that it will be useful, 
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANT ABILITY or FITNESS FOR A PARTICULAR PURPOSE. 
	See the General Public License v3 for more details.
	You should have received a copy of the GNU General Public License
	along with this program. If not, see <http://www.gnu.org/licenses/>.
DISCLAIMER
	The authors of GALAMOST do not guarantee that this program and its 
	derivatives are free from error. In no event shall the copyright 
	holder or contributors be liable for any indirect, incidental, 
	special, exemplary, or consequential loss or damage that results 
	from its use. We also have no responsibility for providing the 
	service of functional extension of this program to general users.
USER OBLIGATION 
	If any results obtained with GALAMOST are published in the scientific 
	literature, the users have an obligation to distribute this program 
	and acknowledge our efforts by citing the paper "Y.-L. Zhu, H. Liu, 
	Z.-W. Li, H.-J. Qian, G. Milano, and Z.-Y. Lu, J. Comput. Chem. 2013,
	34, 2197-2211" in their article.
CORRESPONDENCE
	State Key Laboratory of Polymer Physics and Chemistry,
	Changchun Institute of Applied Chemistry, Chinese Academy of Sciences, China, 
	Dr. You-Liang Zhu, 
	Email: youliangzhu@ciac.ac.cn
*/
//	Maintainer: You-Liang Zhu
#include "BdNvt.cuh"
extern "C" __global__ 
void gpu_bd_nvt_first_step_kernel(Real4* d_pos,
							 Real4* d_vel,
							 Real4* d_force,
							 int3* d_image,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             BoxSize box,
                             Real dt,
							 Real dtsq)
    {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < group_size)
        {
        unsigned int idx = d_group_members[i];
        Real4 pos = d_pos[idx];
        
        Real px = pos.x;
        Real py = pos.y;
        Real pz = pos.z;
        Real pw = pos.w;
        
        Real4 vel = d_vel[idx];
		Real4 accel = d_force[idx];
        Real mass = vel.w;
        accel.x /= mass;
        accel.y /= mass;
        accel.z /= mass;
        
        Real dx = vel.x * dt + 0.5f * accel.x * dtsq;
        Real dy = vel.y * dt + 0.5f * accel.y * dtsq;
        Real dz = vel.z * dt + 0.5f * accel.z * dtsq;
        
        px += dx;
        py += dy;
        pz += dz;

        vel.x += 0.5f * accel.x * dt;
        vel.y += 0.5f * accel.y * dt;
        vel.z += 0.5f * accel.z * dt;

        int3 image = d_image[idx];
		box.wrap(px, py, pz, image);
        
        Real4 pos2;
        pos2.x = px;
        pos2.y = py;
        pos2.z = pz;
        pos2.w = pw;

        d_pos[idx] = pos2;
        d_vel[idx] = vel;
        d_image[idx] = image;
        }
    }
   

hipError_t gpu_bd_nvt_first_step(Real4* d_pos,
							 Real4* d_vel,
							 Real4* d_force,
							 int3* d_image,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             const BoxSize& box,
                             unsigned int block_size,
                             Real dt)
    {
    dim3 grid( (group_size/block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

	Real dtsq = dt*dt;
    gpu_bd_nvt_first_step_kernel<<< grid, threads, block_size * sizeof(Real) >>>(d_pos,
																			 d_vel,	
																			 d_force,
																			 d_image,
                                                                             d_group_members,
                                                                             group_size,
                                                                             box,
                                                                             dt,
																			 dtsq);
    return hipSuccess;
    }


extern "C" __global__ 
void gpu_bd_nvt_second_step_kernel(Real4* d_pos,
								Real4* d_vel,
								Real4* d_force,
								unsigned int *d_group_members,
								unsigned int group_size,
								unsigned int seed,
								Real *d_params,
								Real T,
								Real D,
								Real dt,
								Real dtInv)
 
 {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < group_size)
        {
        unsigned int idx = d_group_members[i];
        Real4 vel = d_vel[idx];
      
        int typ = __real_as_int(d_pos[idx].w);
        Real gamma = d_params[typ];
        Real coeff = sqrt_gala(Real(6.0) * gamma * T * dtInv);       

        Real3 bd_force = ToReal3(0.0, 0.0, 0.0);
        
	    SaruGPU RNG(seed, idx); 
	
        Real randomx = Rondom(-1.0, 1.0);
        Real randomy = Rondom(-1.0, 1.0);
        Real randomz = Rondom(-1.0, 1.0);
        
        bd_force.x = randomx*coeff - gamma*vel.x;
        bd_force.y = randomy*coeff - gamma*vel.y;
        if (D > Real(2.0))
        bd_force.z = randomz*coeff - gamma*vel.z;
        
        Real4 force = d_force[idx];
        Real mass = vel.w;
		
        Real minv = Real(1.0) / mass;
        force.x += bd_force.x;
        force.y += bd_force.y;
        force.z += bd_force.z;

        
        vel.x += Real(0.5) * force.x * minv * dt;
        vel.y += Real(0.5) * force.y * minv * dt;
        vel.z += Real(0.5) * force.z * minv * dt;
        
        d_vel[idx] = vel;
		d_force[idx] = force;
        }

    }

hipError_t gpu_bd_nvt_second_step(Real4* d_pos,
								Real4* d_vel,
								Real4* d_force, 
								unsigned int *d_group_members,
								unsigned int group_size,
								unsigned int seed,
								unsigned int block_size,
								Real *d_params,
								Real T,
								Real D,
								Real dt)
    {
    dim3 grid( (group_size/block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);
	Real dtInv;
	if(dt<1.0e-7)
		dtInv = 0.0;
	else
		dtInv = 1.0/dt;
		
    gpu_bd_nvt_second_step_kernel<<< grid, threads >>>(d_pos,
												d_vel,	
												d_force,
												d_group_members, 
												group_size,
												seed, 
												d_params,
												T,
												D,
												dt,
												dtInv);
    
    return hipSuccess;
    }



